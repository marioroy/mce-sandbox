#include "hip/hip_runtime.h"

// Algorithm3 (parallel range variant).
//   CUDA demonstration by Mario Roy, 2023-09-10
//
// Xuedong Luo:
//   A practical sieve algorithm for finding prime numbers.
//   ACM Volume 32 Issue 3, March 1989, Pages 344-346
//   https://dl.acm.org/doi/pdf/10.1145/62065.62072
//   http://dl.acm.org/citation.cfm?doid=62065.62072
//
//   "Based on the sieve of Eratosthenes, a faster and more compact
//    algorithm is presented for finding all primes between 2 and N.
//    Avoid all composites that have 2 or 3 as one of their prime
//    factors (where i is odd)."
//
//   { 0, 5, 7, 11, 13, ... 3i + 2, 3(i + 1) + 1, ..., N }
//     0, 1, 2,  3,  4, ... list indices (0 is not used)
//
// Build:
//   nvcc -o cudasieve -I../src -O3 -prec-sqrt=true cudasieve.cu -lm
//
// Usage:
//   cudasieve [ N [ N ] [ -p ] ]  default 1 1000
//   cudasieve 100 -p              print primes found
//   cudasieve 1e+10 1.1e+10       count primes found
//   cudasieve 87233720365000000 87233720368547757
//   cudasieve 1e12 1.1e12

#include <hip/hip_runtime.h>
#include <stdint.h>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include "bits.h"

#define CUDACHECK(err) do { cuda_check((err), __FILE__, __LINE__); } while (false)
inline void cuda_check(hipError_t error_code, const char *file, int line)
{
   if (error_code != hipSuccess) {
      fprintf(stderr, "CUDA Error %d: %s. In file '%s' on line %d\n",
         error_code, hipGetErrorString(error_code), file, line);
      fflush(stderr), exit(error_code);
   }
}

byte_t *makeprimes(uint64_t stop)
{
   int64_t q = (int64_t) sqrt((double) stop) / 3;
   int64_t mem_sz = (q + 2 + 7) / 8;
   int64_t i, c = 0, k = 1, t = 2, j, ij;

   byte_t *array = (byte_t *) malloc(mem_sz);
   if (array == NULL) {
      fprintf(stderr, "error: failed to allocate primes array.\n");
      exit(2);
   }
   memset(array, 0xff, mem_sz);
   CLRBIT(array, 0);

   // clear small composites <= q
   for (i = 1; i <= q; i++) {
      k = 3 - k, c += 4 * k * i, j = c;
      ij = 2 * i * (3 - k) + 1, t += 4 * k;
      if (GETBIT(array, i)) {
         while (j <= q) {
            CLRBIT(array, j);
            j += ij, ij = t - ij;
         }
      }
   }

   return array;
}

__global__ static void gpusieve_32(
   byte_t *sieve, const byte_t *is_prime, int64_t step_sz,
   int64_t num_segments, uint64_t start_adj, uint64_t stop, int64_t j_off )
{
   const unsigned int n = __umul24(blockDim.x, blockIdx.x) + threadIdx.x;
   if (n >= num_segments) return;

   static const byte_t unset_bit[8] = {
      (~(1 << 0) & 0xff), (~(1 << 1) & 0xff),
      (~(1 << 2) & 0xff), (~(1 << 3) & 0xff),
      (~(1 << 4) & 0xff), (~(1 << 5) & 0xff),
      (~(1 << 6) & 0xff), (~(1 << 7) & 0xff)
   };

   // account for one-byte padding between segments
   uint32_t s_off = j_off - n * 8, j_off2;
   if (n == 0) { 
      j_off2 = (uint32_t) j_off;
   } else {
      uint64_t low_ = start_adj + step_sz * (n - 1);
      uint64_t high_ = low_ + step_sz - 1;
      if (high_ > stop || high_ < low_) high_ = stop;
      j_off2 = (uint32_t) (high_ / 3);
   }

   // sieve primes
   uint64_t low = start_adj + (step_sz * n);
   uint64_t high = low + step_sz - 1;
   if (high > stop || high < low) high = stop;

   uint32_t c = 0, k = 1, t = 2, j, ij;
   uint32_t n1 = 1, n2 = 2, n3 = 3, n4 = 4;
   uint32_t q = (uint32_t) (sqrt((double) high) / 3);
   uint32_t m = (uint32_t) (high / 3);
   m -= s_off;

   for (uint32_t i = n1; i <= q; i++) {
      k = n3 - k, c += n4 * k * i, j = c;
      ij = n2 * i * (n3 - k) + n1, t += n4 * k;
      if (GETBIT(is_prime, i)) {
         // skip numbers before this segment
         if (j < j_off2) {
            j += (j_off2 - j) / t * t + ij;
            ij = t - ij;
            if (j < j_off2)
               j += ij, ij = t - ij;
         }
         // clear composites
         j -= s_off;
         while (j <= m) {
            sieve[j >> 3] &= unset_bit[j & 7];
            j += ij, ij = t - ij;
         }
      }
   }
}

__global__ static void gpusieve_64(
   byte_t *sieve, const byte_t *is_prime, int64_t step_sz,
   int64_t num_segments, uint64_t start_adj, uint64_t stop, int64_t j_off )
{
   const unsigned int n = __umul24(blockDim.x, blockIdx.x) + threadIdx.x;
   if (n >= num_segments) return;

   static const byte_t unset_bit[8] = {
      (~(1 << 0) & 0xff), (~(1 << 1) & 0xff),
      (~(1 << 2) & 0xff), (~(1 << 3) & 0xff),
      (~(1 << 4) & 0xff), (~(1 << 5) & 0xff),
      (~(1 << 6) & 0xff), (~(1 << 7) & 0xff)
   };

   // account for one-byte padding between segments
   int64_t s_off = j_off - n * 8, j_off2;
   if (n == 0) { 
      j_off2 = j_off;
   } else {
      uint64_t low_ = start_adj + step_sz * (n - 1);
      uint64_t high_ = low_ + step_sz - 1;
      if (high_ > stop || high_ < low_) high_ = stop;
      j_off2 = high_ / 3;
   }

   // sieve primes
   uint64_t low = start_adj + (step_sz * n);
   uint64_t high = low + step_sz - 1;
   if (high > stop || high < low) high = stop;

   int64_t c = 0, j; uint32_t k = 1, t = 2, ij;
   uint32_t n1 = 1, n2 = 2, n3 = 3, n4 = 4;
   uint32_t q = (uint32_t) (sqrt((double) high) / 3);
   int64_t m = high / 3;
   m -= s_off;

   for (int32_t i = n1; i <= q; i++) {
      k = n3 - k, c += n4 * k * i, j = c;
      ij = n2 * i * (n3 - k) + n1, t += n4 * k;
      if (GETBIT(is_prime, i)) {
         // skip numbers before this segment
         if (j < j_off2) {
            j += (j_off2 - j) / t * t + ij;
            ij = t - ij;
            if (j < j_off2)
               j += ij, ij = t - ij;
         }
         // clear composites
         j -= s_off;
         while (j <= m) {
            sieve[j >> 3] &= unset_bit[j & 7];
            j += ij, ij = t - ij;
         }
      }
   }
}

void cudasieve(uint64_t start, uint64_t stop, int print_flag)
{
   // adjust start to a multiple of 6; then subtract 6 and add 1
   uint64_t start_adj = (start > 5)
      ? start - (start % 6) - 6 + 1
      : 1;

   int64_t bsize, step_sz;

   if (stop < 1e11)
      { bsize =  4, step_sz = 39600; }
   else if (stop < 1e14)
      { bsize =  8, step_sz = 39600; }
   else if (stop < 1e16)
      { bsize = 16, step_sz = 39600 * 3; }
   else
      { bsize = 32, step_sz = 39600 * 5; }

   if      ( stop >= 1e+19 ) { step_sz *= 80; }
   else if ( stop >= 1e+18 ) { step_sz *= 70; }
   else if ( stop >= 1e+17 ) { step_sz *= 60; }
   else if ( stop >= 1e+16 ) { step_sz *= 50; }
   else if ( stop >= 1e+15 ) { step_sz *= 40; }
   else if ( stop >= 1e+14 ) { step_sz *= 30; }
   else if ( stop >= 1e+13 ) { step_sz *= 20; }
   else if ( stop >= 1e+12 ) { step_sz *= 10; }

   int64_t num_segments = (stop - start_adj + step_sz) / step_sz;
   byte_t *is_prime = makeprimes(stop);
   int64_t count = 0;

   int64_t M = (stop - start_adj + (stop & 1)) / 3;
   uint64_t n_off = start_adj - 1;
   int64_t j_off = n_off / 3;
   int64_t mem_sz = (M + 2 + 7) / 8 + (num_segments - 1);
   byte_t *sieve;

   sieve = (byte_t *) malloc(mem_sz);
   if (sieve == NULL) {
      fprintf(stderr, "error: failed to allocate sieve array.\n");
      exit(2);
   }
   memset(sieve, 0xff, mem_sz);
   CLRBIT(sieve, 0);

   // clear bits less than start
   if (n_off + ((3 * 1 + 1) | 1) < start) {
      CLRBIT(sieve, 1);
      if (n_off + ((3 * 2 + 1) | 1) < start)
         CLRBIT(sieve, 2);
   }

   // clear bits greater than stop
   int64_t i = (mem_sz - (num_segments - 1)) * 8 - (M + 2);
   while (i) {
      CLRBIT(sieve, mem_sz * 8 - i);
      i--;
   }
   if (n_off + ((3 * (M + 1) + 1) | 1) > stop) {
      CLRBIT(sieve, M + 1 + (num_segments - 1) * 8);
      if (n_off + ((3 * M + 1) | 1) > stop)
         CLRBIT(sieve, M + (num_segments - 1) * 8);
   }

   // create MM list; clear one-byte padding between segments
   int64_t *MM = (int64_t *) malloc(num_segments * sizeof(int64_t));
   int64_t off = 0;

   for (int64_t n = 0; n < num_segments - 1; n++) {
      uint64_t low = start_adj + (step_sz * n);
      uint64_t high = low + step_sz - 1;
      if (high > stop || high < low) high = stop;
      int64_t m = high / 3;
      MM[n] = m - j_off;
      for (int i = 1; i <= 8; i++)
         CLRBIT(sieve, m - j_off + i + off);
      off += 8;
   }

   MM[num_segments - 1] = M + 2;
   int64_t gsize = num_segments / bsize + (num_segments % bsize ? 1 : 0);

   // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
   // CUDA BEGIN //////////////////////////////////////////////////////////////
   // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

   // init CUDA, change integer on a multi-GPU system
   CUDACHECK(hipSetDevice(0));
 
   int64_t q = (int64_t) sqrt((double) stop) / 3;
   int64_t mem_sz_p = (q + 2 + 7) / 8;

   byte_t *d_is_prime = NULL;  // size mem_sz_p
   byte_t *d_sieve = NULL;     // size mem_sz
   float kernel_time;

   // allocate memory on the device
   CUDACHECK(hipMalloc((void**) &d_is_prime, sizeof(byte_t) * mem_sz_p));
   CUDACHECK(hipMalloc((void**) &d_sieve, sizeof(byte_t) * mem_sz));

   // copy -> device
   CUDACHECK(hipMemcpy(d_is_prime, is_prime, sizeof(byte_t) * mem_sz_p, hipMemcpyHostToDevice));
   CUDACHECK(hipMemcpy(d_sieve, sieve, sizeof(byte_t) * mem_sz, hipMemcpyHostToDevice));

   // run the kernel
   hipEvent_t kernel_start, kernel_stop;
   hipEventCreate(&kernel_start);
   hipEventCreate(&kernel_stop);
   hipEventRecord(kernel_start, 0);

   if (stop <= 12700000000)  // 1.27e10
      gpusieve_32<<<gsize, bsize, 0>>>( d_sieve, d_is_prime,
         step_sz, num_segments, start_adj, stop, j_off );
   else
      gpusieve_64<<<gsize, bsize, 0>>>( d_sieve, d_is_prime,
         step_sz, num_segments, start_adj, stop, j_off );

   hipDeviceSynchronize();
   hipEventRecord(kernel_stop, 0);
   hipEventSynchronize(kernel_stop);
   hipEventElapsedTime(&kernel_time, kernel_start, kernel_stop);
   hipEventDestroy(kernel_start);
   hipEventDestroy(kernel_stop);

   // copy -> host
   CUDACHECK(hipMemcpy(sieve, d_sieve, sizeof(byte_t) * mem_sz, hipMemcpyDeviceToHost));

   // release memory
   hipFree(d_sieve);
   hipFree(d_is_prime);

   // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
   // CUDA END ////////////////////////////////////////////////////////////////
   // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

   free((void *) is_prime); is_prime = NULL;

   if (start <= 2 && stop >= 2) count++;
   if (start <= 3 && stop >= 3) count++;

   count += popcount(sieve, mem_sz);

   if (print_flag) {
      if (start <= 2 && stop >= 2) printf("2\n");
      if (start <= 3 && stop >= 3) printf("3\n");
      int64_t off = 0, num = MM[0], ind = 0;
      for (i = 1; i <= M; i += 2) {
         if (i >= num)
            off += 8, num = MM[++ind];
         if (GETBIT(sieve, i + off))
            printf("%lu\n", n_off + (3 * i + 2));
         if (GETBIT(sieve, i + 1 + off))
            printf("%lu\n", n_off + (3 * (i + 1) + 1));
      }
   }

   free((void *) MM); MM = NULL;
   free((void *) sieve); sieve = NULL;

   fprintf(stderr, "Primes found: %ld\n", count);
   fprintf(stderr, " Kernel time: %0.3lf\n", kernel_time / 1000);
}

int main(int argc, char** argv)
{
   // find primes in range, inclusively
   uint64_t start = 1, stop = 1000;
   int print_flag = 0;

   // check for print option (last option specified)
   if (argc > 1 && strcmp(argv[argc-1], "-p") == 0) {
      print_flag = 1;
      argc--;
   }

   if (argc > 2) {
      start = (uint64_t) strtold(argv[1], NULL);
      stop  = (uint64_t) strtold(argv[2], NULL);
   }
   else if (argc > 1) {
      stop  = (uint64_t) strtold(argv[1], NULL);
   }

   if (stop > 0 && stop >= start) {
      if (stop - start > 1e+11) {
         fprintf(stderr, "Range distance exceeds 1e+11 (~4GB).\n");
         return 1;
      }

      clock_t tstart = clock();
      cudasieve(start, stop, print_flag);
      clock_t tend = clock();

      double elapsed_time = ((double) (tend - tstart)) / CLOCKS_PER_SEC;
      fprintf(stderr, "  Total time: %0.3lf\n", elapsed_time);
   }

   return 0;
}

